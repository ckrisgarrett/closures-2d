#include "hip/hip_runtime.h"
/*
 File:   fobj_cuda.cu
 Author: Kris Garrett
 Date:   June 7, 2012
 
 Mimics fobj.cpp
 
 h CANNOT BE NULL
 This is important.  fobj should call this routine only when
 H != NULL
 
 g can be NULL
*/

#include "fobj_cuda.h"
#include <stdio.h>
#include "../../utils.h"


// The device variables.
// These are allocated on the CUDA cards in fobj_initialize.
static double **wDev;
static double **pDev;
static double **alphaDev;
static double **hDev;
static double **tempDev;

// Pinned memory and streams to use CUDA Streams.
static double **alphaPinned;
static double **hPinned;
static double **pPinned;
static hipStream_t *stream;

// Used to determine if CUDA should be used.
static int numGpu;
static int numThreadsPerGpu;


static void checkError(hipError_t error, int line)
{
    if(error != hipSuccess)
        printf("%d: %s\n", line, hipGetErrorString(error));
}


/*
    Fills in the array e^(alpha^T p).
    The array is of size quadrature points.
*/
__global__
void kernel_fillTempArrayMn(int nm, int nq, double *alpha, double *w, double *p,
                            double *tempArray)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k < nq)
    {
        tempArray[k] = 0;
        for(int i = 0; i < nm; i++)
        {
            tempArray[k] = tempArray[k] + alpha[i] * p[k * nm + i];
        }
        tempArray[k] = exp(tempArray[k]) * w[k];
    }
}


/*
    Calculates the Hessian for the Mn algorithm.
*/
__global__
void kernel_setHMn(int nm, int nq, double *tempArray, double *p, 
                   double *h)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / nm;
    int j = index % nm;
    if(index < nm * nm)
    {
        h[i * nm + j] = 0;
        
        // Do the integral part of the calculation.
        for(int k = 0; k < nq; k++)
        {
            h[i * nm + j] = h[i * nm + j] + 
                tempArray[k] * p[k * nm + i] * p[k * nm + j];
        }
    }
}


/*
    Calculates alpha^T p.
*/
__global__
void kernel_fillTempArrayPPn(int nm, int nq, double *alpha, double *p,
                             double *tempArray)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k < nq)
    {
        tempArray[k] = 0;
        for(int i = 0; i < nm; i++)
        {
            tempArray[k] = tempArray[k] + alpha[i] * p[k * nm + i];
        }
    }
}


/*
    Calculates the Hessian for the PPn algorithm.
*/
__global__
void kernel_setHPPn(int nm, int nq, double *tempArray, double *w, double *p, double delta, 
                    double *h)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / nm;
    int j = index % nm;
    if(index < nm * nm)
    {
        h[i * nm + j] = 0;
        
        // Do the integral part of the calculation.
        for(int k = 0; k < nq; k++)
        {
            double t_k = tempArray[k];
            h[i * nm + j] = h[i * nm + j] + 0.5 * w[k] * p[k * nm + i] * p[k * nm + j] * 
                           (1.0 + t_k / sqrt(t_k * t_k + 4.0 * delta));
        }
    }
}


/*
    Determines if CUDA should be used.
*/
extern "C"
int fobjUseCuda(int thread)
{
    // Return true
    if(thread < numGpu * numThreadsPerGpu)
        return 1;

    // Return false
    return 0;
}


/*
    Initialization that must be done in serial.
*/
extern "C"
void fobjCudaInitSerial(int numCudaCards, int numThreadsPerCudaCard)
{
    // Check for enough cuda cards and allocate memory for arrays.
    int numCards;
    checkError(hipGetDeviceCount(&numCards), __LINE__);
    if(numCards < numCudaCards)
    {
        printf("fobj_cuda: Not enough CUDA cards in machine.\n");
        printf("fobj_cuda: Number of Cards: %d\n", numCards);
        printf("fobj_cuda: numCudaCards defined in the input deck: %d\n", numCudaCards);
        utils_abort();
    }

    wDev = (double**)malloc(numCudaCards * numThreadsPerCudaCard * sizeof(double*));
    pDev = (double**)malloc(numCudaCards * numThreadsPerCudaCard * sizeof(double*));
    alphaDev = (double**)malloc(numCudaCards * numThreadsPerCudaCard * sizeof(double*));
    hDev = (double**)malloc(numCudaCards * numThreadsPerCudaCard * sizeof(double*));
    tempDev = (double**)malloc(numCudaCards * numThreadsPerCudaCard * sizeof(double*));
    alphaPinned = (double**)malloc(numCudaCards * numThreadsPerCudaCard * sizeof(double*));
    hPinned = (double**)malloc(numCudaCards * numThreadsPerCudaCard * sizeof(double*));
    pPinned = (double**)malloc(numCudaCards * numThreadsPerCudaCard * sizeof(double*));
    stream = (hipStream_t*)malloc(numCudaCards * numThreadsPerCudaCard * sizeof(hipStream_t));
    
    numGpu = numCudaCards;
    numThreadsPerGpu = numThreadsPerCudaCard;
}




/*
    Initialization that must be done in parallel for openmp.
*/
extern "C"
void fobjCudaInitParallel(int nm, int nq, double *w, int thread)
{
    // Allocate memory on card and set quadrature.
    if(thread < numGpu * numThreadsPerGpu)
    {
        int gpu = thread % numGpu;
        checkError(hipSetDevice(gpu), __LINE__);
        checkError(hipStreamCreate(&stream[thread]), __LINE__);

        checkError(hipMalloc((void**)&wDev[thread], nq * sizeof(double)), __LINE__);
        checkError(hipMalloc((void**)&pDev[thread], nq * nm * sizeof(double)), __LINE__);
        checkError(hipMalloc((void**)&alphaDev[thread], nm * sizeof(double)), __LINE__);
        checkError(hipMalloc((void**)&hDev[thread],     nm * nm * sizeof(double)), __LINE__);
        checkError(hipMalloc((void**)&tempDev[thread],  nq * sizeof(double)), __LINE__);
        
        checkError(hipMemcpy(wDev[thread], w, nq * sizeof(double), hipMemcpyHostToDevice), __LINE__);

        checkError(hipHostAlloc((void**)&alphaPinned[thread], nm * sizeof(double), 
            hipHostMallocPortable), __LINE__);
        checkError(hipHostAlloc((void**)&hPinned[thread], nm * nm * sizeof(double), 
            hipHostMallocPortable), __LINE__);
        checkError(hipHostAlloc((void**)&pPinned[thread], nq * nm * sizeof(double), 
            hipHostMallocPortable), __LINE__);
    }
}


/*
    fobj replacement using CUDA for the Mn algorithm.
*/
extern "C"
double fobjMnCuda(int nm, int nq, double *alpha, double *u, double *p, int thread, 
                  double *g, double *h)
{
    int numThreadsPerBlock = 256;
    int numBlocks = (nq-1) / numThreadsPerBlock + 1;
    int numThreadsPerBlockH = 256;
    int numBlocksH = (nm*nm-1) / numThreadsPerBlockH + 1;
    
    // Set alpha on card and solve for H.
    memcpy(alphaPinned[thread], alpha, nm * sizeof(double));
    memcpy(pPinned[thread], p, nm * nq * sizeof(double));
    checkError(hipMemcpyAsync(alphaDev[thread], alphaPinned[thread], nm * sizeof(double), 
               hipMemcpyHostToDevice, stream[thread]), __LINE__);
    checkError(hipMemcpyAsync(pDev[thread], pPinned[thread], nm * nq * sizeof(double), 
               hipMemcpyHostToDevice, stream[thread]), __LINE__);
    
    kernel_fillTempArrayMn<<<numBlocks, numThreadsPerBlock, 0, stream[thread]>>>(nm, nq, 
                          alphaDev[thread], wDev[thread], pDev[thread], 
                          tempDev[thread]);
    kernel_setHMn<<<numBlocksH, numThreadsPerBlockH, 0, stream[thread]>>>(nm, nq, 
                 tempDev[thread], pDev[thread], hDev[thread]);
    
    checkError(hipMemcpyAsync(hPinned[thread], hDev[thread], nm * nm * sizeof(double), 
               hipMemcpyDeviceToHost, stream[thread]), __LINE__);
    checkError(hipStreamSynchronize(stream[thread]), __LINE__);
    memcpy(h, hPinned[thread], nm * nm * sizeof(double));

    
    // Get g.
    double p00 = p[0];
    if(g != NULL);
    {
        for(int k = 0; k < nm; k++)
            g[k] = h[k] / p00 - u[k];
    }
    
    // Get f.
    double f = h[0] / (p00 * p00);
    
    // f = f - alpha^T u
    for(int k = 0; k < nm; k++)
        f = f - alpha[k] * u[k];

    return f;
}


/*
    Calculates f and g for the PPn algorithm.
*/
static
double getfgPPn(int nm, int nq, double *alpha, double *u, double *w, double *p, 
                double delta, double *g)
{
    double f = 0;
    for(int k = 0; k < nm; k++)
        g[k] = 0.0;

    // Integrate.
    for(int q = 0; q < nq; q++)
    {
        // Compute temp = alpha^T * p.
        double temp = 0;
        for(int i = 0; i < nm; i++)
        {
            temp = temp + alpha[i] * p[q*nm+i];
        }

        // Compute temp2 = sqrt((alpha^t * p)^2 + 4 * delta).
        double temp2 = sqrt(temp * temp + 4 * delta);

        // Compute temp3 = 0.5 alpha^T * p + 0.5 sqrt((alpha^T * p)^2 + 4 * delta).
        double temp3 = 0.5 * (temp + temp2);

        // Compute f.
        f = f + w[q] * (0.5 * temp3 * temp3 - delta + delta * log(temp3));

        // Compute g.
        double temp4 = w[q] * temp3;
        for(int i = 0; i < nm; i++)
            g[i] = g[i] + p[q*nq+i] * temp4;
    }

    // Finish the computations not requiring integration.
    for(int i = 0; i < nm; i++)
    {
        f = f - alpha[i] * u[i];

        if(g != NULL)
            g[i] = g[i] - u[i];
    }

    return f;
}


/*
    fobj replacement using CUDA for the PPn algorithm.
*/
extern "C"
double fobjPPnCuda(int nm, int nq, double *alpha, double *u, double *w, double *p, 
                   double delta, int thread, double *g, double *h)
{
    int numThreadsPerBlock = 96;
    int numBlocks = (nq-1) / numThreadsPerBlock + 1;
    int numThreadsPerBlockH = 96;
    int numBlocksH = (nm*nm-1) / numThreadsPerBlockH + 1;
    
    // Set alpha on card and solve for H.
    memcpy(alphaPinned[thread], alpha, nm * sizeof(double));
    memcpy(pPinned[thread], p, nm * nq * sizeof(double));
    hipMemcpyAsync(alphaDev[thread], alphaPinned[thread], nm * sizeof(double), 
               hipMemcpyHostToDevice, stream[thread]);
    hipMemcpyAsync(pDev[thread], pPinned[thread], nm * nq * sizeof(double), 
               hipMemcpyHostToDevice, stream[thread]);
    
    kernel_fillTempArrayPPn<<<numBlocks, numThreadsPerBlock, 0, stream[thread]>>>
        (nm, nq, alphaDev[thread], pDev[thread], tempDev[thread]);
    kernel_setHPPn<<<numBlocksH, numThreadsPerBlockH, 0, stream[thread]>>>
        (nm, nq, tempDev[thread], wDev[thread], pDev[thread], delta, hDev[thread]);
    
    double f = getfgPPn(nm, nq, alpha, u, w, p, delta, g);
    
    hipMemcpyAsync(hPinned[thread], hDev[thread], nm * nm * sizeof(double), 
               hipMemcpyDeviceToHost, stream[thread]);
    hipStreamSynchronize(stream[thread]);
    memcpy(h, hPinned[thread], nm * nm * sizeof(double));
    

    return f;
}

