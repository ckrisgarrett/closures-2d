#include "hip/hip_runtime.h"
/*
 File:   solve_flux_cuda.cu
 Author: Kris Garrett
 Date:   April 19, 2013
*/

#include "solve_flux_cuda.h"
#include "../utils.h"


// GPU device variables
static double *alphaDev;
static double *pDev;
static double *ansatzDev;
static double *wDev;
static double *xiDev;
static double *etaDev;
static double *fluxDev;
static double *fluxTempDev;


/*
    Initialization for solveFlux_cuda.

    nx:  size of grid in x-direction
    ny:  size of grid in y-direction
    nm:  number of moments
    w:   quadrature weights
    p:   moments at quadrature points
    xi:  xi values at quadrature points
    eta: eta values at quadrature points
*/
extern "C"
void solveFluxInit_cuda(int nx, int ny, int nm, int nm2, int nq, double *w, double *p, 
                        double *xi, double *eta)
{
    hipMalloc((void**)&alphaDev,  nx * ny * nm * sizeof(double));
    hipMalloc((void**)&pDev,      nq * nm * sizeof(double));
    hipMalloc((void**)&ansatzDev, nx * ny * sizeof(double));
    hipMalloc((void**)&wDev,      nq * sizeof(double));
    hipMalloc((void**)&xiDev,     nq * sizeof(double));
    hipMalloc((void**)&etaDev,    nq * sizeof(double));
    hipMalloc((void**)&fluxDev,   nx * ny * nm * sizeof(double));
    hipMalloc((void**)&fluxTempDev,   nx * ny * sizeof(double));
    
    // Must do copy one row at a time since nm2 may be bigger than number of moments
    // for Clebsch-Gordan.
    double *pSmall = (double*)malloc(nq * nm * sizeof(double));
    for(int q = 0; q < nq; q++)
    {
        memcpy(&pSmall[q * nm], &p[q * nm2], nm * sizeof(double));
    }
    hipMemcpy(pDev,   pSmall,  nq * nm * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(wDev,   w,       nq * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(xiDev,  xi,      nq * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(etaDev, eta,     nq * sizeof(double), hipMemcpyHostToDevice);
    
    free(pSmall);
}


/*
    Helper functions to calculate slopes.
*/
__device__
double minmod(double x, double y)
{
    return SIGN(1.0,x) * MAX(0.0, MIN(fabs(x), y * SIGN(1.0, x)));
}

__device__
double slopefit(double left, double center, double right, double theta)
{
    return minmod(theta*(right-center), minmod(0.5*(right-left), theta*(center-left)));
}


/*
    Computes the ansatz at quadrature index q for the entire grid.
    
    n:      total size of the grid
    nm:     number of moments
    q:      quadrature index
    alpha:  grid of alpha vectors
    p:      moments
    ansatz: grid of ansatzes
*/
/*__global__
void kernel_computeAnsatz(int n, int nm, int q, double *alpha, double *p, double *ansatz)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(index < n)
    {
        double *alpha_i = &alpha[index * nm];
        double kinetic = 0.0;
        for(int k = 0; k < nm; k++)
        {
            kinetic += alpha_i[k] * p[q * nm + k];
        }
        ansatz[index] = exp(kinetic);
    }
}*/


__global__
void kernel_computeAnsatz(int n, int nm, int q, double *alpha, double *p, double *ansatz)
{
    __shared__ double temp[16][16];
    int space_index = blockIdx.x * blockDim.x + threadIdx.y;
    
    
    temp[threadIdx.y][threadIdx.x] = 0.0;
    __syncthreads();
    
    
    if(space_index < n)
    {
        for(int batch_index = 0; batch_index < nm; batch_index += 16)
        {
            int k = batch_index + threadIdx.x;
            
            if(k < nm)
                temp[threadIdx.y][threadIdx.x] += alpha[space_index * nm + k] * p[q*nm+k];
        }
        __syncthreads();
        
        if(threadIdx.y == 0)
        {
            double ans = 0.0;
            for(int i = 0; i < 16; i++)
            {
                ans += temp[threadIdx.x][i];
            }
            ans = exp(ans);
            ansatz[blockIdx.x * blockDim.x + threadIdx.x] = ans;
        }
    }
}


/*
    Solve flux on GPU.

    nx:     size of grid in x-direction
    ny:     size of grid in y-direction
    q:      quadrature index
    nm:     number of moments
    theta:  value for slopefit
    xi:     xi at quadrature points
    eta:    eta at quadrature points
    w:      quadrature weights
    p:      moments at quadrature points
    ansatz: grid of ansatzes
    flux:   grid of fluxes
*/
/*__global__
void kernel_solveFlux(int nx, int ny, int q, int nm, double theta, double dx, double dy, 
                      double *xi, double *eta, double *w, double *p, double *ansatz, double *flux)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / ny;
    int j = index % ny;
    
    if( (index < nx * ny) && (i > 2) && (i < nx - 2) && (j > 2) && (j < ny - 2) )
    {
        double flux1 = 0;
        double flux2 = 0;
        double xi_q = xi[q];
        double eta_q = eta[q];
        double w_q = w[q];
        
        if(xi_q > 0)
        {
            double k1 = ansatz[(i-2) * ny + j];
            double k2 = ansatz[(i-1) * ny + j];
            double k3 = ansatz[(i)   * ny + j];
            double k4 = ansatz[(i+1) * ny + j];
            
            flux1 = k3 + 0.5 * slopefit(k2, k3, k4, theta) - 
                k2 - 0.5 * slopefit(k1, k2, k3, theta);
            flux1 = flux1 * xi_q * w_q;
        }
        else
        {
            double k1 = ansatz[(i-1) * ny + j];
            double k2 = ansatz[(i)   * ny + j];
            double k3 = ansatz[(i+1) * ny + j];
            double k4 = ansatz[(i+2) * ny + j];
            
            flux1 = k3 - 0.5 * slopefit(k2, k3, k4, theta) - 
                k2 + 0.5 * slopefit(k1, k2, k3, theta);
            flux1 = flux1 * xi_q * w_q;
        }
        if(eta_q > 0)
        {
            double k1 = ansatz[i * ny + (j-2)];
            double k2 = ansatz[i * ny + (j-1)];
            double k3 = ansatz[i * ny + (j)];
            double k4 = ansatz[i * ny + (j+1)];
            
            flux2 = k3 + 0.5 * slopefit(k2, k3, k4, theta) - 
                k2 - 0.5 * slopefit(k1, k2, k3, theta);
            flux2 = flux2 * eta_q * w_q;
        }
        else
        {
            double k1 = ansatz[i * ny + (j-1)];
            double k2 = ansatz[i * ny + (j)];
            double k3 = ansatz[i * ny + (j+1)];
            double k4 = ansatz[i * ny + (j+2)];
            
            flux2 = k3 - 0.5 * slopefit(k2, k3, k4, theta) - 
                k2 + 0.5 * slopefit(k1, k2, k3, theta);
            flux2 = flux2 * eta_q * w_q;
        }
        
        
        for(int k = 0; k < nm; k++)
        {
            flux[index * nm + k] += p[q * nm + k] * (flux1 / dx + flux2 / dy);
        }
    }
}*/


__global__
void kernel_solveFluxTemp(int nx, int ny, int q, int nm, double theta, double dx, double dy, 
                      double *xi, double *eta, double *w, double *p, double *ansatz, double *fluxTemp)
{
    __shared__ double ansatz_shared[20][20];
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    
    
    ansatz_shared[threadIdx.y][threadIdx.x] = ansatz[(i-2) * ny + (j-2)];
    if(threadIdx.y < 4)
        ansatz_shared[threadIdx.y+16][threadIdx.x] = ansatz[(i-2+16) * ny + (j-2)];
    if(threadIdx.x < 4)
        ansatz_shared[threadIdx.y][threadIdx.x+16] = ansatz[(i-2) * ny + (j-2+16)];
    if(threadIdx.x < 4 && threadIdx.y < 4)
        ansatz_shared[threadIdx.y+16][threadIdx.x+16] = ansatz[(i-2+16) * ny + (j-2+16)];
    __syncthreads();
    
    
    if( (i > 2) && (i < nx - 2) && (j > 2) && (j < ny - 2) )
    {
        double flux1 = 0;
        double flux2 = 0;
        double xi_q = xi[q];
        double eta_q = eta[q];
        double w_q = w[q];
        
        if(xi_q > 0)
        {
            double k1 = ansatz_shared[threadIdx.y+2-2][threadIdx.x+2];
            double k2 = ansatz_shared[threadIdx.y+2-1][threadIdx.x+2];
            double k3 = ansatz_shared[threadIdx.y+2][threadIdx.x+2];
            double k4 = ansatz_shared[threadIdx.y+2+1][threadIdx.x+2];
            
            flux1 = k3 + 0.5 * slopefit(k2, k3, k4, theta) - 
                k2 - 0.5 * slopefit(k1, k2, k3, theta);
            flux1 = flux1 * xi_q * w_q;
        }
        else
        {
            double k1 = ansatz_shared[threadIdx.y+2-1][threadIdx.x+2];
            double k2 = ansatz_shared[threadIdx.y+2][threadIdx.x+2];
            double k3 = ansatz_shared[threadIdx.y+2+1][threadIdx.x+2];
            double k4 = ansatz_shared[threadIdx.y+2+2][threadIdx.x+2];
            
            flux1 = k3 - 0.5 * slopefit(k2, k3, k4, theta) - 
                k2 + 0.5 * slopefit(k1, k2, k3, theta);
            flux1 = flux1 * xi_q * w_q;
        }
        if(eta_q > 0)
        {
            double k1 = ansatz_shared[threadIdx.y+2][threadIdx.x+2-2];
            double k2 = ansatz_shared[threadIdx.y+2][threadIdx.x+2-1];
            double k3 = ansatz_shared[threadIdx.y+2][threadIdx.x+2];
            double k4 = ansatz_shared[threadIdx.y+2][threadIdx.x+2+1];
            
            flux2 = k3 + 0.5 * slopefit(k2, k3, k4, theta) - 
                k2 - 0.5 * slopefit(k1, k2, k3, theta);
            flux2 = flux2 * eta_q * w_q;
        }
        else
        {
            double k1 = ansatz_shared[threadIdx.y+2][threadIdx.x+2-1];
            double k2 = ansatz_shared[threadIdx.y+2][threadIdx.x+2];
            double k3 = ansatz_shared[threadIdx.y+2][threadIdx.x+2+1];
            double k4 = ansatz_shared[threadIdx.y+2][threadIdx.x+2+2];
            
            flux2 = k3 - 0.5 * slopefit(k2, k3, k4, theta) - 
                k2 + 0.5 * slopefit(k1, k2, k3, theta);
            flux2 = flux2 * eta_q * w_q;
        }
        
        fluxTemp[i*ny+j] = flux1 / dx + flux2 / dy;
        /*for(int k = 0; k < nm; k++)
        {
            flux[(i*ny+j) * nm + k] += p[q * nm + k] * (flux1 / dx + flux2 / dy);
        }*/
    }
}


__global__
void kernel_solveFlux(int nx, int ny, int nm, int q, double *fluxTemp, double *p, double *flux)
{
    int space_index = blockIdx.x * blockDim.x + threadIdx.y;
    int i = space_index / ny;
    int j = space_index % ny;
    
    if( (i > 2) && (i < nx - 2) && (j > 2) && (j < ny - 2) )
    {
        for(int batch_index = 0; batch_index < nm; batch_index += 16)
        {
            int k = threadIdx.x + batch_index;
            if(k < nm)
                flux[space_index * nm + k] += p[q * nm + k] * fluxTemp[space_index];
        }
    }
}


/*
    Solve flux on GPU driver.

    nx:    size of grid in x-direction
    ny:    size of grid in y-direction
    nm:    number of moments
    q:     quadrature index
    theta: value for slopefit
    alpha: grid of alpha vectors
    flux:  grid of fluxes
*/
extern "C"
void solveFlux_cuda(int nx, int ny, int nm, int nq, double theta, double dx, double dy, 
                    double *alpha, double *flux)
{
    int n = nx * ny;
    
    dim3 nt1(16,16);
    int nb1 = (n-1) / 16 + 1;
    dim3 nt2(16,16);
    dim3 nb2((ny-1)/16+1, (nx-1)/16+1);
    
    hipMemcpy(alphaDev, alpha, n * nm * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(fluxDev, 0, n * nm * sizeof(double));
    for(int q = 0; q < nq; q++)
    {
        kernel_computeAnsatz<<<nb1, nt1>>>(n, nm, q, alphaDev, pDev, ansatzDev);
        kernel_solveFluxTemp<<<nb2, nt2>>>(nx, ny, q, nm, theta, dx, dy, 
                                       xiDev, etaDev, wDev, pDev, ansatzDev, fluxTempDev);
        kernel_solveFlux<<<nb1, nt1>>>(nx, ny, nm, q, fluxTempDev, pDev, fluxDev);
    }
    hipMemcpy(flux, fluxDev, n * nm * sizeof(double), hipMemcpyDeviceToHost);
}

